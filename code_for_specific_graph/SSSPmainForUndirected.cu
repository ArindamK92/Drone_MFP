#include <stdio.h>
#include "all_structure_undir.cuh"
#include "gpuFunctions_undir.cuh"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include<vector>
#include <chrono>
#include <algorithm>
#include "cuCompactor.cuh"
#include "supportingFunctions.cu"


#define THREADS_PER_BLOCK 1024 //we can change it

using namespace std;
using namespace std::chrono;



/*
1st arg: original graph file name
2nd arg: no. of nodes
3rd arg: no. of edges
4th arg: input SSSP file name
5th arg: change edges file name
6th arg: drone start vertex
7th arg: destination vertex
8th arg: payload (0 or 7)
9th arg: output file name
****main commands to run****
nvcc -o op_main CudaSSSPmain.cu
./op_main original_graph_file_name number_of_nodes number_of_edges input_SSSP_file_name change_edge_file_name
*/
int main(int argc, char* argv[]) {

	int nodes, edges, deviceId, numberOfSMs;
	int no_of_movement = 0;
	hipError_t cudaStatus;
	char* graphFile = argv[1];
	nodes = atoi(argv[2]);
	edges = atoi(argv[3]);
	char* inputSSSPfile = argv[4];
	char* changeEdgesFile = argv[5];
	char* outFile = argv[9]; //output file

	//Drone related
	int currentLoc = 0; //drone's current location. considering single drone single depot.
	int nextLoc; //drone's current location
	int droneStartLoc = atoi(argv[6]); //drone start vertex
	int destination = atoi(argv[7]); //destination vertex 
	int payload = atoi(argv[8]); //payload
	int cost = 0; //total cost for travel
	int* traversed;
	traversed = (int*)calloc(nodes, sizeof(int));
	int ws[4] = {0,5,10,15}; //keep the choices here//change if taking different dataset
	int wd[5] = {180,135,90,45,0}; //keep the choices here//change if taking different dataset
	
	int oldRand = 0, oldRand2 = 0;
	
	
	
	


	while (currentLoc != destination && no_of_movement < 20) {

		if (no_of_movement > 0)
		{
			graphFile = "nextGraph.txt";
			inputSSSPfile = "nextSSSP.txt";
			changeEdgesFile = "nextEffectiveChangeEdges.txt"; //have to vary this randomly
		}



		int totalInsertion = 0;
		bool zeroDelFlag = false, zeroInsFlag = false;
		vector<ColWtList> AdjList; //stores input graph in 2D adjacency list
		vector<ColWt> AdjListFull; //Row-major implementation of adjacency list (1D)
		ColWt* AdjListFull_device; //1D array in GPU to store Row-major implementation of adjacency list 
		int* AdjListTracker_device; //1D array to track offset for each node's adjacency list
		vector<changeEdge> allChange_Ins, allChange_Del;
		changeEdge* allChange_Ins_device; //stores all change edges marked for insertion in GPU
		changeEdge* allChange_Del_device; //stores all change edges marked for deletion in GPU
		int* counter_del;
		int* affectedNodeList_del;
		int* updatedAffectedNodeList_del;
		int* updated_counter_del;
		vector<ColList> SSSPTreeAdjList;
		int* SSSPTreeAdjListTracker;
		vector<int> SSSPTreeAdjListFull;
		RT_Vertex* SSSP;
		int* SSSPTreeAdjListFull_device;
		int* SSSPTreeAdjListTracker_device;
		vector<int> hop;
		int* d_hop;





		//Get gpu device id and number of SMs
		hipGetDevice(&deviceId);
		hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);
		size_t  numberOfBlocks = 32 * numberOfSMs;

		//Read Original input graph
		AdjList.resize(nodes);
		int* AdjListTracker = (int*)malloc((nodes + 1) * sizeof(int));//we take nodes +1 to store the start ptr of the first row
		read_graphEdges(AdjList, graphFile, &nodes);
		
		

		//Read change edges input
		readin_changes(changeEdgesFile, allChange_Ins, allChange_Del, AdjList, totalInsertion);
		int totalChangeEdges_Ins = allChange_Ins.size();
		if (totalChangeEdges_Ins == 0) {
			zeroInsFlag = true;
		}
		int totalChangeEdges_Del = allChange_Del.size();
		if (totalChangeEdges_Del == 0) {
			zeroDelFlag = true;
		}

		//Transfer input graph, changed edges to GPU and set memory advices
		transfer_data_to_GPU(AdjList, AdjListTracker, AdjListFull, AdjListFull_device,
			nodes, edges, totalInsertion, AdjListTracker_device, zeroInsFlag,
			allChange_Ins, allChange_Ins_device, totalChangeEdges_Ins,
			deviceId, totalChangeEdges_Del, zeroDelFlag, allChange_Del_device,
			counter_del, affectedNodeList_del, updatedAffectedNodeList_del, updated_counter_del, allChange_Del, numberOfBlocks);


		//Read input SSSP Tree and storing on unified memory
		read_and_transfer_input_SSSPtree_to_GPU(inputSSSPfile, SSSPTreeAdjList, SSSPTreeAdjListTracker, SSSPTreeAdjListFull,
			SSSP, nodes, edges, SSSPTreeAdjListFull_device, SSSPTreeAdjListTracker_device, hop, deviceId, d_hop);


		//Initialize supporting variables
		int* change = 0;
		cudaStatus = hipMallocManaged(&change, sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed at change structure");
		}
		int* affectedNodeList;
		cudaStatus = hipMallocManaged(&affectedNodeList, nodes * sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed at affectedNodeList structure");
		}
		int* counter = 0;
		cudaStatus = hipMallocManaged(&counter, sizeof(int));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc failed at counter structure");
		}
		int* updatedAffectedNodeList_all;
		hipMallocManaged(&updatedAffectedNodeList_all, nodes * sizeof(int));
		int* updated_counter_all = 0;
		hipMallocManaged(&updated_counter_all, sizeof(int));





		//**process change edges**
		auto startTimeDelEdge = high_resolution_clock::now(); //Time calculation start
		//Process del edges
		if (zeroDelFlag != true) {

			deleteEdge << < numberOfBlocks, THREADS_PER_BLOCK >> > (allChange_Del_device, SSSP, totalChangeEdges_Del, AdjListFull_device, AdjListTracker_device);
			hipDeviceSynchronize();
		}
		auto stopTimeDelEdge = high_resolution_clock::now();//Time calculation ends
		auto durationDelEdge = duration_cast<microseconds>(stopTimeDelEdge - startTimeDelEdge);// duration calculation
		//cout << "**Time taken for processing deleted edges: "<< float(durationDelEdge.count()) / 1000 << " milliseconds**" << endl;





		//Process ins edges
		auto startTimeinsertEdge = high_resolution_clock::now();
		if (zeroInsFlag != true) {

			insertEdge << < numberOfBlocks, THREADS_PER_BLOCK >> > (allChange_Ins_device, SSSP, totalChangeEdges_Ins, AdjListFull_device, AdjListTracker_device);
			hipDeviceSynchronize();
		}
		auto stopTimeinsertEdge = high_resolution_clock::now();//Time calculation ends
		auto durationinsertEdge = duration_cast<microseconds>(stopTimeinsertEdge - startTimeinsertEdge);// duration calculation
		//cout << "**Time taken for processing inserted Edges: "<< float(durationinsertEdge.count()) / 1000 << " milliseconds**" << endl;

		//new code
		//int* Q_array = (int*)malloc((nodes) * sizeof(int));
		//vector<int> Del_Affected_array;

		//auto startTime1 = high_resolution_clock::now();
		//*counter_del = cuCompactor::compact<RT_Vertex, int>(SSSP, affectedNodeList_del, nodes, predicate(), THREADS_PER_BLOCK);
		///*for (int i = 0; i < *counter_del; i++) {
		//	printf("%d::", affectedNodeList_del[i]);
		//}*/
		////hipDeviceSynchronize();
		//for (int i = 0; i < *counter_del; i++) {
		//	int x = affectedNodeList_del[i];
		//	Del_Affected_array.push_back(x);
		//}
		////printf("test 2");
		//int j = 0;
		//while (j < Del_Affected_array.size())
		//{
		//	int x = Del_Affected_array[j];
		//	if (SSSPTreeAdjList.at(x).size() > 0) {
		//		Del_Affected_array.insert(std::end(Del_Affected_array), std::begin(SSSPTreeAdjList.at(x)), std::end(SSSPTreeAdjList.at(x)));
		//	}
		//	
		//	j++;
		//}
		//auto stopTime1 = high_resolution_clock::now();//Time calculation ends
		//auto durationin1 = duration_cast<microseconds>(stopTime1 - startTime1);// duration calculation
		//cout << "**Time taken for creating Del_Affected_array: "
		//	<< float(durationin1.count()) / 1000 << " milliseconds**" << endl;
		//cout << "size of Del_Affected_array:" << j << endl;
		//for (int i = 0; i < Del_Affected_array.size(); i++) {
		//	printf("%d::", Del_Affected_array[i]);
		//}
		//new code 


		//**make the subtree under deletion affected vertices disconnected (make wt = inf)
		auto startTimeupdateNeighbors_del = high_resolution_clock::now();
		if (zeroDelFlag != true) {
			*counter_del = cuCompactor::compact<RT_Vertex, int>(SSSP, affectedNodeList_del, nodes, predicate(), THREADS_PER_BLOCK);
			*change = 1;
			while (*change > 0) {
				*change = 0;
				updateNeighbors_del << <numberOfBlocks, THREADS_PER_BLOCK >> >
					(SSSP, updated_counter_del, updatedAffectedNodeList_del, affectedNodeList_del, counter_del, SSSPTreeAdjListFull_device, SSSPTreeAdjListTracker_device, change);
				*counter_del = cuCompactor::compact<RT_Vertex, int>(SSSP, affectedNodeList_del, nodes, predicate(), THREADS_PER_BLOCK);
				//printf("number of elements in the compacted list: %d\n", *counter_del);
				//hipDeviceSynchronize();//not required as hipMalloc/hipFree perform heavy-weight synchronizations. cuCompactor::compact uses both in it.
			}
		}
		hipFree(SSSPTreeAdjListFull_device); //we can free memory at the end if we have enough GPU memory. That will decrease some time
		hipFree(SSSPTreeAdjListTracker);

		auto stopTimeupdateNeighbors_del = high_resolution_clock::now();//Time calculation ends
		auto durationupdateNeighbors_del = duration_cast<microseconds>(stopTimeupdateNeighbors_del - startTimeupdateNeighbors_del);// duration calculation
		//cout << "**Time taken for updateNeighbors_del: "<< float(durationupdateNeighbors_del.count()) / 1000 << " milliseconds**" << endl;



		//**Update neighbors and connect disconnected vertices with main SSSP tree**
		auto startTimeupdateNeighbors = high_resolution_clock::now();

		//collect all vertices where update value > 0
		*counter = cuCompactor::compact<RT_Vertex, int>(SSSP, affectedNodeList, nodes, predicate2(), THREADS_PER_BLOCK);

		*change = 1;
		while (*change == 1) {
			*change = 0;
			updateNeighbors << <(*counter / THREADS_PER_BLOCK) + 1, THREADS_PER_BLOCK >> > (SSSP, counter, affectedNodeList, AdjListFull_device, AdjListTracker_device, change);
			*counter = cuCompactor::compact<RT_Vertex, int>(SSSP, affectedNodeList, nodes, predicate2(), THREADS_PER_BLOCK);
			//hipDeviceSynchronize(); //not required as hipMalloc/hipFree perform heavy-weight synchronizations. cuCompactor::compact uses both in it.
		}

		auto stopTimeupdateNeighbors = high_resolution_clock::now();//Time calculation ends
		auto durationupdateNeighbors = duration_cast<microseconds>(stopTimeupdateNeighbors - startTimeupdateNeighbors);// duration calculation
		//cout << "**Time taken for updateNeighbors: "<< float(durationupdateNeighbors.count()) / 1000 << " milliseconds**" << endl;






		cout << "****Total Time taken for SSSP update: "
			<< (float(durationDelEdge.count()) + float(durationupdateNeighbors_del.count()) + float(durationinsertEdge.count()) + float(durationupdateNeighbors.count())) / 1000 << " milliseconds****" << endl;



		//cout << "Total affected nodes by Delete edge only: " << totalAffectedNodes_del << endl;

		//print node parent distance
		//cout << "from GPU: \n[";
		//printSSSP << <1, 1 >> > (SSSP, nodes);
		//hipDeviceSynchronize();
		//int x;
		//if (nodes < 40) {
		//	x = nodes;
		//}
		//else {
		//	x = 40;
		//}
		////cout << "from CPU: \n[";
		//for (int i = 0; i < x; i++) {
		//	cout << i << " " << SSSP[i].Parent << " " << SSSP[i].Dist << endl;
		//	//cout << i << ":" << SSSP[i].Dist << " ";
		//}


		//****Print next move****
		traversed[currentLoc] = 1;
		int parent = -1;
		int y = destination;
		cout << "print path" << endl;
		cout << y;
		while (y != 0) {
			parent = SSSP[y].Parent;
			cout << "<- " << parent << "(" << SSSP[y].Dist - SSSP[parent].Dist << ")"; //test

			if (parent == currentLoc)
			{
				cost = cost + SSSP[y].Dist - SSSP[parent].Dist;
				cout << "\nNext move: " << parent << "to" << y << " cost upto this: " << cost << endl;
				nextLoc = y;
				
			}

			y = parent;
			//cout << i << ":" << SSSP[i].Dist << " ";

		}



		//****print sssp tree in file. format: vertex parent distance****
		ofstream myfile("nextSSSP.txt");
		if (myfile.is_open())
		{
			for (int i = 0; i < nodes; i++) {
				string line = to_string(i) + " " + to_string(SSSP[i].Parent) + " " + to_string(SSSP[i].Dist) + "\n";
				myfile << line;
			}
			myfile.close();
		}
		else cout << "Unable to open nextSSSP.txt file";

		//****print current graph in file. format: vertex1 vertex2 weight****
		ofstream myfile2("nextGraph.txt");
		int nextEdges = 0;
		if (myfile2.is_open())
		{
			for (int i = 0; i < nodes; i++) {
				for (int j = AdjListTracker[i]; j < AdjListTracker[i + 1]; j++) {
					int myn = AdjListFull_device[j].col;
					int mywt = AdjListFull_device[j].wt;
					if (mywt < 0) { continue; } //if mywt = -1, that means edge was deleted
					if (myn < i) { continue; } //avoid printing "b a w1" if "a b w1" is already printed
					string line = to_string(i) + " " + to_string(myn) + " " + to_string(mywt) + "\n";
					myfile2 << line;
					nextEdges++;
				}
			}
			myfile2.close();
		}
		else cout << "Unable to open nextGraph.txt file";


		//****choose and print next effective change edges****
		srand(time(NULL));
		int random1 = rand() % 4; //choice for ws
		int random2 = rand() % 5; //choice for wd
		if(oldRand == random1)
		{
			random1 = (random1 + 1)%4;
		}
		if(oldRand2 == random2)
		{
			random2 = (random2 + 1)%5; //adding 1 just to create another number
		}
		oldRand = random1;
		oldRand2 = random2;
		string ceFileName = "TATA_p" + to_string(payload) + "_ws" + to_string(ws[random1]) +"_wd"+to_string(wd[random2]) + ".txt";
		//int filename_length = ceFileName.length();
		cout<<"next changeEdgeFile::"<<ceFileName<<endl;
		const char *char_fileName = ceFileName.c_str();
		
		ofstream myfile3("nextEffectiveChangeEdges.txt");
		if (myfile3.is_open())
		{
			
			//insert new edges
			FILE* delE_file;
			char line[128];
			//delE_file = fopen("nextChangeEdges.txt", "r"); //select the next Del E***implement a random choice fn***
			delE_file = fopen(char_fileName, "r"); //select the next Del E
			while (fgets(line, 128, delE_file) != NULL)
			{
				int n1, n2, wt;
				changeEdge cE;
				sscanf(line, "%d %d %d", &n1, &n2, &wt);
				//Add change edge in effective change edge only when none of the endpoint is traversed
				if (traversed[n1] == 0 && traversed[n2] == 0)
				{
					int flag1 = 0;
					//****delete edge (u,n,wt) when drone moves from u to v****
					for (int j = AdjListTracker[n1]; j < AdjListTracker[n1 + 1]; j++) {
						int myn = AdjListFull_device[j].col;
						int mywt = AdjListFull_device[j].wt;
						if (mywt < 0) { continue; } //if mywt = -1, that means edge was deleted
						if (myn == n2)
						{
							if (mywt != wt) {
								string line1 = to_string(n1) + " " + to_string(myn) + " " + to_string(mywt) + " " + to_string(0) + "\n"; //delete previous edge
								myfile3 << line1;
								//cout << line1 << endl;
								string line2 = to_string(n1) + " " + to_string(n2) + " " + to_string(wt) + " " + to_string(1) + "\n"; //insert new edge
								myfile3 << line2;
								//cout << line2 << endl;
							}
							
							flag1 = 1;
							break;
						}
					}
					if (flag1 == 0)
					{
						string line2 = to_string(n1) + " " + to_string(n2) + " " + to_string(wt) + " " + to_string(1) + "\n"; //insert new edge
						myfile3 << line2;
						//cout << line2 << endl;
					}
				}
			}
			fclose(delE_file);

			//****delete edge (u,n,wt) when drone moves from u to v****
			for (int j = AdjListTracker[currentLoc]; j < AdjListTracker[currentLoc + 1]; j++) {
				int myn = AdjListFull_device[j].col;
				int mywt = AdjListFull_device[j].wt;
				if (mywt < 0) { continue; } //if mywt = -1, that means edge was deleted
				if (myn == nextLoc) { continue; } //skip as nextLoc is v and (u,v) should be 0
				if (traversed[myn] == 1) { continue; }
				string line4 = to_string(currentLoc) + " " + to_string(myn) + " " + to_string(mywt) + " " + to_string(0) + "\n";
				myfile3 << line4;
			}
			//insert edge (u,v,0) when drone moves from u to v
			string line1 = to_string(currentLoc) + " " + to_string(nextLoc) + " " + to_string(0) + " " + to_string(1) + "\n";
			myfile3 << line1;

			myfile3.close();
		}
		else cout << "Unable to open nextEffectiveChangeEdges.txt file";



		//prepare for next iteration
		currentLoc = nextLoc; //prepare current location for next iteration
		edges = nextEdges;
		no_of_movement++;


		if (zeroDelFlag != true) {
			hipFree(affectedNodeList_del);
			hipFree(updatedAffectedNodeList_del);
			hipFree(counter_del);
			hipFree(updated_counter_del);
			hipFree(allChange_Del_device);

		}
		if (zeroInsFlag != true) {
			hipFree(allChange_Ins_device);
		}
		hipFree(change);
		hipFree(affectedNodeList);
		hipFree(counter);
		hipFree(AdjListFull_device);
		hipFree(AdjListTracker_device);

		hipFree(SSSP);
		hipFree(d_hop); //try to free this at some earlier place


	}

	std::ofstream ofs;
	ofs.open (outFile, std::ofstream::out | std::ofstream::app);
	ofs << droneStartLoc << " " << destination << " " << cost <<"\n";
	ofs.close();

	return 0;
}